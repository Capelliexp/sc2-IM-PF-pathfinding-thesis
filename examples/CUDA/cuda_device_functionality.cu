#include "hip/hip_runtime.h"
#pragma once

#include "../examples/CUDA/cuda_header.cuh"
#include "../examples/CUDA/cuda_device_utility.cu"

/*
PF Todo:
* Quad-tree for units
* Compare simultaneous global write vs non-simultaneous
* Compare different block sizes & dimensions
*/

__global__ void DeviceRepellingPFGeneration(Entity* device_unit_list_pointer, int nr_of_units, hipPitchedPtr device_map_ground, hipPitchedPtr device_map_air) {
	extern __shared__ Entity unit_list_s[];

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int id_block = threadIdx.x + threadIdx.y * blockDim.x;
	int id_global = x + y * blockDim.x;

	//move unit list to shared memory
	if (id_block < nr_of_units) unit_list_s[id_block] = device_unit_list_pointer[id_block];

	//cull threads outside of tex
	if (x > MAP_X_R || y > MAP_Y_R) return;

	__syncthreads();

	//add upp all fields affecting the owned coord to ground_charge and air_charge
	float ground_charge = 0;
	float air_charge = 0;
	float dist = 0;
	for (int i = 0; i < nr_of_units; ++i) {
		UnitInfoDevice unit = device_unit_lookup[unit_list_s[i].id];
		float range_sub = unit.range;

		if ((dist = (FloatDistance(unit_list_s[i].pos.x, unit_list_s[i].pos.y, x, y) + 0.0001)) < range_sub) {
			ground_charge += ((range_sub / dist) * unit.can_attack_ground * unit_list_s[i].enemy);
			air_charge += ((range_sub / dist) * unit.can_attack_air * unit_list_s[i].enemy);
		}
	}

	//__syncthreads();
	
	//write ground_charge and air_charge to global memory in owned coord
	((float*)(((char*)device_map_ground.ptr) + y * device_map_ground.pitch))[x] = ground_charge;
	((float*)(((char*)device_map_air.ptr) + y * device_map_ground.pitch))[x] = air_charge;
}

__global__ void DeviceAttractingPFGeneration(Entity* device_unit_list_pointer, int nr_of_units, int owner_type_id, hipPitchedPtr device_map){

}

__global__ void DeviceGroundIMGeneration(IntPoint2D destination, hipPitchedPtr device_map, hipPitchedPtr dynamic_map, hipPitchedPtr static_map) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int id_block = threadIdx.x + threadIdx.y * blockDim.x;
	int id_global = x + y * blockDim.x;

	//cull threads outside of tex
	if (x > MAP_X_R || y > MAP_Y_R) return;

}

__global__ void DeviceAirIMGeneration(IntPoint2D destination, hipPitchedPtr device_map) {

}