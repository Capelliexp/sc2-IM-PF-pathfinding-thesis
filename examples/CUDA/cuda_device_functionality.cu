#include "hip/hip_runtime.h"
#pragma once

#include "../examples/CUDA/cuda_header.cuh"
#include "../examples/CUDA/cuda_device_utility.cu"

/*
PF Todo:
* Quad-tree for units
* Compare simultaneous global write vs non-simultaneous
* Compare different block sizes & dimensions
*/

__global__ void DeviceRepellingPFGeneration(Entity* device_unit_list_pointer, int nr_of_units, hipPitchedPtr device_map_ground, hipPitchedPtr device_map_air) {
	extern __shared__ Entity unit_list_s[];

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int id_block = threadIdx.x + threadIdx.y * blockDim.x;
	int id_global = x + y * blockDim.x;

	//move unit list to shared memory
	if (id_block < nr_of_units) unit_list_s[id_block] = device_unit_list_pointer[id_block];

	//cull threads outside of tex
	if (x >= MAP_X_R || y >= MAP_Y_R) return;

	__syncthreads();

	//add upp all fields affecting the owned coord to ground_charge and air_charge
	float ground_charge = 0;
	float air_charge = 0;
	float dist = 0;
	for (int i = 0; i < nr_of_units; ++i) {
		UnitInfoDevice unit = device_unit_lookup[unit_list_s[i].id];
		float range_sub = unit.range;

		if ((dist = (FloatDistance(unit_list_s[i].pos.x, unit_list_s[i].pos.y, x, y) + 0.0001)) < range_sub) {
			ground_charge += ((range_sub / dist) * unit.can_attack_ground * unit_list_s[i].enemy);
			air_charge += ((range_sub / dist) * unit.can_attack_air * unit_list_s[i].enemy);
		}
	}

	//__syncthreads();
	
	//write ground_charge and air_charge to global memory in owned coord
	((float*)(((char*)device_map_ground.ptr) + y * device_map_ground.pitch))[x] = ground_charge;
	((float*)(((char*)device_map_air.ptr) + y * device_map_ground.pitch))[x] = air_charge;
}

__global__ void DeviceAttractingPFGeneration(Entity* device_unit_list_pointer, int nr_of_units, int owner_type_id, hipPitchedPtr device_map){

}

__global__ void DeviceGroundIMGeneration(IntPoint2D destination, hipPitchedPtr device_map, hipPitchedPtr dynamic_map, list_double_entry* global_memory_im_list_storage) {
	int block_size = blockDim.x*blockDim.y;
	int grid_size = (gridDim.x * blockDim.x) * (gridDim.y * blockDim.y);

	int id_block = blockIdx.x + blockIdx.y * gridDim.x;
	int original_x = threadIdx.x + blockIdx.x * blockDim.x;
	int original_y = threadIdx.y + blockIdx.y * blockDim.y;
	int original_id = threadIdx.x + id_block * block_size + threadIdx.y * blockDim.x;

	int start_id = (original_id + (original_id % block_size) * block_size) % grid_size;
	int x = start_id % (MAP_X_R);
	int y = start_id / (float)(MAP_X_R);

	//int start_id = original_id;
	//int x = original_x;
	//int y = original_y;

	if (x >= MAP_X_R || y >= MAP_Y_R) return; //cull threads outside of tex
	if (GetBoolMapValue(dynamic_map, x, y) == 0) {	//cull threads in terrain
		((float*)(((char*)device_map.ptr) + y * device_map.pitch))[x] = -2;
		return;
	}

	list_double_entry* open_list = (list_double_entry*)malloc(3000 * sizeof(list_double_entry));
	list_double_entry* closed_list = (list_double_entry*)malloc(3000 * sizeof(list_double_entry));
	//int open_list[100];
	//list_double_entry closed_list[100];
	//array max size: 75 000
	//list_double_entry* open_list = &global_memory_im_list_storage[start_id * (150000/2)];
	//list_double_entry* closed_list = &global_memory_im_list_storage[start_id * (150000/2) + (256000000/2)];
	int open_list_it = 0, closed_list_it = 0, open_list_size = 3000, closed_list_size = 3000;

	//__syncthreads();

	if (open_list == NULL || closed_list == NULL) {
		printf("Device heap limit to low for lists\n");
		return;
	}

	open_list[0] = { start_id , -1 };
	//++open_list_it;
	open_list_it = 1;

	int size_check_counter = 0;
	for (int step_iterator = 0; step_iterator < MAP_SIZE_R; ++step_iterator) {
		//find the next cell to expand
		float closest_distance_found = 9999999;
		float curr_node_dist = 0;
		list_double_entry closest_entry;
		int closest_coord_found = -1;
		list_double_entry entry;

		for (int i = 0; i < open_list_it; ++i) {
			entry = open_list[i];
			if (entry.node != -1) {
				if ((curr_node_dist = FloatDistanceFromIDRelative(entry.node, destination)) < closest_distance_found) {
					closest_distance_found = curr_node_dist;
					closest_coord_found = i;
					closest_entry.node = entry.node;
					closest_entry.backtrack_iterator = entry.backtrack_iterator;
				}
			}
		}

		if (closest_coord_found == -1) {//open list is empty and no path to the destination is found, RIP
			((float*)(((char*)device_map.ptr) + y * device_map.pitch))[x] = -1;
			return;
		}

		//add the expanded coord to the closed list
		closed_list[closed_list_it] = { closest_entry.node, closest_entry.backtrack_iterator };
		++closed_list_it;

		int pos_x = closest_entry.node % MAP_X_R;
		int pos_y = closest_entry.node / (float)(MAP_X_R);

		if ((pos_x == destination.x) && (pos_y == destination.y)) {	//destination has been found! HYPE
			Backtrack(device_map, closed_list, closed_list_it - 1);
			free(open_list);
			free(closed_list);
			return;
		}

		//expand the size of the open and closed list if necessary
		if (size_check_counter == 30) {
			size_check_counter = 0;
			if ((open_list_size - open_list_it) < 200) {
				list_double_entry* open_list_new = (list_double_entry*)malloc(open_list_size * 2 * sizeof(list_double_entry));
				memcpy(open_list_new, open_list, open_list_size);
				open_list_size *= 2;
				free(open_list);
				open_list = open_list_new;
			}
			if ((closed_list_size - closed_list_it) < 200) {
				list_double_entry* closed_list_new = (list_double_entry*)malloc(closed_list_size * 2 * sizeof(list_double_entry));
				memcpy(closed_list_new, closed_list, closed_list_size);
				closed_list_size *= 2;
				free(closed_list);
				closed_list = closed_list_new;
			}
		}

		//add the expanded nodes neighbours to the open list
		short_coord neighbour_coords[4];
		neighbour_coords[0] = { pos_x, pos_y - 1 };
		neighbour_coords[1] = { pos_x - 1, pos_y };
		neighbour_coords[2] = { pos_x + 1, pos_y };
		neighbour_coords[3] = { pos_x, pos_y + 1 };

		int new_open_list_entries = 0;
		for (int i = 0; i < 4; ++i) {
			int coord_global = neighbour_coords[i].x + (neighbour_coords[i].y * MAP_X_R);

			if (GetBoolMapValue(dynamic_map, coord_global) != 0) {	//coord not in terrain
				if (IDInList(coord_global, open_list, open_list_it) == -1 && IDInList(coord_global, closed_list, closed_list_it) == -1) {	//coord not already in open or closed list
					open_list[open_list_it + new_open_list_entries] = { coord_global, (closed_list_it - 1) };
					++new_open_list_entries;
				}
			}
		}
		open_list_it += new_open_list_entries;

		open_list[closest_coord_found].node = -1;	//mark expanded node as invalid in the open list
		++size_check_counter;
	}
}

__device__ void Backtrack(hipPitchedPtr device_map, list_double_entry* closed_list, int start_it) {
	list_double_entry node = closed_list[start_it];

	for (int loop_count = 1; loop_count < MAP_SIZE_R + 1; ++loop_count) {
		int x = node.node % MAP_X_R;
		int y = node.node / (float)(MAP_X_R);

		((float*)(((char*)device_map.ptr) + y * device_map.pitch))[x] = loop_count;

		if (node.backtrack_iterator == -1) return;
		node = closed_list[node.backtrack_iterator];
	}
}


__global__ void DeviceAirIMGeneration(IntPoint2D destination, hipPitchedPtr device_map) {

}