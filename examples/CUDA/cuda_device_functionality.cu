#include "hip/hip_runtime.h"
#pragma once

#include "../examples/CUDA/cuda_header.cuh"
#include "../examples/CUDA/cuda_device_utility.cu"

__global__ void DeviceRepellingPFGeneration(Entity* device_unit_list_pointer, int nr_of_units, hipPitchedPtr device_map_ground, hipPitchedPtr device_map_air) {
	extern __shared__ Entity unit_list_s[];
	memset(unit_list_s, 0, (nr_of_units * sizeof(Entity)) + ((32 * sizeof(Entity)) - ((nr_of_units * sizeof(Entity))) % 32));

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int id_block = threadIdx.x + threadIdx.y * blockDim.x;

	//move unit list to shared memory
	if (id_block < nr_of_units) unit_list_s[id_block] = device_unit_list_pointer[id_block];

	if (x >= MAP_X_R || y >= MAP_Y_R || x < 0 || y < 0) {	//return if start tex is out of bounds 
		return;
	}

	const int register_list_size = 32;
	Entity register_list[register_list_size];

	__syncthreads();

	float ground_charge = 0;
	float air_charge = 0;
	float largest_ground_charge = 0;
	float largest_air_charge = 0;

	int nr_of_slice_loops = ((float)((int)(((float)nr_of_units / (float)register_list_size) - 0.00001)) + 1);
	for (int slice = 0; slice < nr_of_slice_loops; ++slice) {
		for (int i = 0; i < register_list_size; ++i) {
			register_list[i] = unit_list_s[i + (slice * register_list_size)];
		}

		for (int i = 0; i < register_list_size; ++i) {
			if (register_list[i].id < 1 || register_list[i].id > 120) break;	//break if we reach end of list

			UnitInfoDevice unit = device_unit_lookup[register_list[i].id];
			float range_sub = fmaxf(unit.range, 3) + 2;
			float dist = (FloatDistance((int)register_list[i].pos.x, (int)register_list[i].pos.y, x, y) + 0.0001);

			if (register_list[i].enemy) {	//avoid enemies
				if (dist < range_sub) {
					float curr_ground_charge = ((range_sub / dist) * unit.can_attack_ground) + 50;
					float curr_air_charge = ((range_sub / dist) * unit.can_attack_air) + 50;
					if (curr_ground_charge > largest_ground_charge) largest_ground_charge = curr_ground_charge;
					if (curr_air_charge > largest_air_charge) largest_air_charge = curr_air_charge;
				}
			}
			else {	//avoid friendlies
				int res = 1 - (int)dist + (int)(unit.radius + 0.5);
				if (res > 0) {
					ground_charge += (res / 2.f) * !(unit.is_flying);
					air_charge += (res / 2.f) * unit.is_flying;
				}
			}
		}
	}

	//write ground_charge and air_charge to global memory in owned coord
	((float*)(((char*)device_map_ground.ptr) + y * device_map_ground.pitch))[x] = ground_charge + largest_ground_charge;
	((float*)(((char*)device_map_air.ptr) + y * device_map_ground.pitch))[x] = air_charge + largest_air_charge;
}

__global__ void DeviceLargeRepellingPFGeneration(Entity* device_unit_list_pointer, int nr_of_units, hipPitchedPtr device_map_ground, hipPitchedPtr device_map_air) {
	extern __shared__ Entity unit_list_s[];
	memset(unit_list_s, 0, (nr_of_units * sizeof(Entity)) + ((32 * sizeof(Entity)) - ((nr_of_units * sizeof(Entity))) % 32));

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int id_block = threadIdx.x + threadIdx.y * blockDim.x;
	int id_global = x + y * blockDim.x;

	//move unit list to shared memory
	if (id_block < nr_of_units) unit_list_s[id_block] = device_unit_list_pointer[id_block];

	if (x >= MAP_X_R || y >= MAP_Y_R || x < 0 || y < 0) {	//return if start tex is out of bounds 
		return;
	}

	const int register_list_size = 32;
	Entity register_list[register_list_size];

	__syncthreads();

	float max_value = 100;
	float falloff = 2;
	float range_sub = 14;
	float ground_charge = 0;
	float air_charge = 0;
	float largest_ground_charge = 0;
	float largest_air_charge = 0;

	int nr_of_slice_loops = ((float)((int)(((float)nr_of_units / (float)register_list_size) - 0.00001)) + 1);
	for (int slice = 0; slice < nr_of_slice_loops; ++slice) {
		for (int i = 0; i < register_list_size; ++i) {
			register_list[i] = unit_list_s[i + (slice * register_list_size)];
		}

		for (int i = 0; i < register_list_size; ++i) {
			if (register_list[i].id < 1 || register_list[i].id > 120) break;

			UnitInfoDevice unit = device_unit_lookup[register_list[i].id];
			float dist = (FloatDistance((int)register_list[i].pos.x, (int)register_list[i].pos.y, x, y) + 0.0001);

			if (register_list[i].enemy) {	//avoid enemies
				if (dist < range_sub) {
					float curr_ground_charge = ((max_value - (falloff * dist)) * unit.can_attack_ground);
					float curr_air_charge = ((max_value - (falloff * dist)) * unit.can_attack_air);
					if (curr_ground_charge > largest_ground_charge) largest_ground_charge = curr_ground_charge;
					if (curr_air_charge > largest_air_charge) largest_air_charge = curr_air_charge;
				}
			}
			else {	//avoid friendlies
				int res = 1 - (int)dist + (int)(unit.radius + 0.5);
				if (res > 0) {
					ground_charge += (res / 2.f) * !(unit.is_flying);
					air_charge += (res / 2.f) * unit.is_flying;
				}
			}
		}
	}

	//write ground_charge and air_charge to global memory in owned coord
	((float*)(((char*)device_map_ground.ptr) + y * device_map_ground.pitch))[x] = ground_charge + largest_ground_charge;
	((float*)(((char*)device_map_air.ptr) + y * device_map_ground.pitch))[x] = air_charge + largest_air_charge;
}

__global__ void DeviceAttractingPFGeneration(Entity* device_unit_list_pointer, int nr_of_units, int owner_type_id, hipPitchedPtr device_map){
	extern __shared__ Entity unit_list_s[];
	memset(unit_list_s, 0, (nr_of_units * sizeof(Entity)) + ((32 * sizeof(Entity)) - ((nr_of_units * sizeof(Entity))) % 32));

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int id_block = threadIdx.x + threadIdx.y * blockDim.x;

	//move unit list to shared memory
	if (id_block < nr_of_units) unit_list_s[id_block] = device_unit_list_pointer[id_block];

	if (x >= MAP_X_R || y >= MAP_Y_R || x < 0 || y < 0) {	//return if start tex is out of bounds 
		return;
	}

	const int register_list_size = 32;
	Entity register_list[register_list_size];

	__syncthreads();

	UnitInfoDevice self_info = device_unit_lookup[owner_type_id];

	float tot_charge = 0;
	UnitInfoDevice other_info;
	Entity other_entity;

	int nr_of_slice_loops = ((float)((int)(((float)nr_of_units / (float)register_list_size) - 0.00001)) + 1);
	for (int slice = 0; slice < nr_of_slice_loops; ++slice) {
		for (int i = 0; i < register_list_size; ++i) {
			if (i + slice * register_list_size >= nr_of_units) break;
			register_list[i] = unit_list_s[i + (slice * register_list_size)];
		}

		for (int i = 0; i < register_list_size; ++i) {
			if (register_list[i].id < 1 || register_list[i].id > 120) break;

			other_info = device_unit_lookup[register_list[i].id];
			other_entity = register_list[i];

			float dist = (FloatDistance((int)other_entity.pos.x, (int)other_entity.pos.y, x, y) + 0.0001);
			bool self_can_attack_other = (other_info.is_flying && self_info.can_attack_air) || (!other_info.is_flying && self_info.can_attack_ground);

			if (other_entity.enemy) {	//attack enemy
				if (self_can_attack_other) {	//can attack unit
					if (self_info.range < 1.1) {	//self is melee
						if (dist < 10) {	//attack enemy
							tot_charge -= 10 / dist;
						}
					}
					else {	//self is ranged
						float range_diff = self_info.range - other_info.range;
						if (range_diff >= 0) {	//self more range than other
							if (dist <= (self_info.range + (self_info.radius /*+ 1*/))) {	//avoid area close to enemy
								tot_charge += 40 - (dist * 2);
							}
							else if (dist < self_info.range * 1.2 || dist < 10) {	//attack enemy
								tot_charge -= 10 - (dist);
							}
						}
						else {	//attack other with larger range than self
							tot_charge -= 10 / dist;
						}
					}
				}
			}
			else {	//avoid friend
				if (self_info.is_flying == other_info.is_flying) {
					int res = 1 - (int)dist + (int)(other_info.radius + 0.5);
					if (res > 0) {
						tot_charge += (res / 2.f);
					}
				}
			}
		}
	}

	((float*)(((char*)device_map.ptr) + y * device_map.pitch))[x] = tot_charge;
}

//Maximum number of 32-bit registers per thread block: 64k
//Maximum number of 32-bit registers per thread: 255
//Maximum amount of shared memory per multiprocessor: 48kB (49152B)
//Maximum amount of shared memory per thread block: 48kB (49152B)
//Number of shared memory banks: 32
//Amount of local memory per thread: 512KB
//Constant memory size: 64KB

__global__ void DeviceGroundIMGeneration(IntPoint2D destination, hipPitchedPtr device_map, hipPitchedPtr dynamic_map) {
	int block_size = blockDim.x * blockDim.y;
	int grid_size = (gridDim.x * blockDim.x) * (gridDim.y * blockDim.y);
	int grid_thread_width = gridDim.x * blockDim.x;

	int thread_id_in_block = threadIdx.x + (threadIdx.y * blockDim.x);
	int id_block = blockIdx.x + (blockIdx.y * gridDim.x);
	int original_x = threadIdx.x + (blockIdx.x * blockDim.x);
	int original_y = threadIdx.y + (blockIdx.y * blockDim.y);
	int original_id = threadIdx.x + (id_block * block_size) + (threadIdx.y * blockDim.x);

	//thread spreading
	//int start_id = (original_id + (original_id % block_size) * block_size) % grid_size;
	//int x = (start_id % MAP_X_R);
	//int y = (start_id / (float)MAP_X_R);

	//original
	integer start_id = (integer)original_id;
	integer x = (start_id % grid_thread_width);
	integer y = (start_id / (float)grid_thread_width);

	IntPoint2D debug_coord = {10, 49};
	bool debug = false;

	//if (debug && debug_coord.x == x && debug_coord.y == y) printf(" \n");

	if (destination.x >= MAP_X_R || destination.y >= MAP_Y_R) {	//return if destination is out of bounds
		((float*)(((char*)device_map.ptr) + y * device_map.pitch))[x] = -1;
		if (x == 0 && y == 0) printf("CUDA PRINT: destination out of bounds\n");
		return;
	}

	if (GetBoolMapValue(dynamic_map, destination.x, destination.y) == 0) {	//return if destination is unreachable
		((float*)(((char*)device_map.ptr) + y * device_map.pitch))[x] = -1;
		if (x == 0 && y == 0) printf("CUDA PRINT: destination unreachable\n");
		return;
	}

	if (x >= MAP_X_R || y >= MAP_Y_R || x < 0 || y < 0) {	//return if start tex is out of bounds 
		return;
	}

	if (GetBoolMapValue(dynamic_map, x, y) == 0) {	//return if start tex is in terrain
		((float*)(((char*)device_map.ptr) + y * device_map.pitch))[x] = -2;
		return;
	}

	//CREATE REGISTER ARRAY
	const int register_list_size = 32;
	node register_list[register_list_size];

	//CREATE SHARED ARRAY
	const int mem_per_SM = 47616;	//normal: 49152, 1536 (384 per block) reserved for block intra-communication
	const int mem_per_block = mem_per_SM / 4;
	const int nodes_per_block = mem_per_block / sizeof(node);	// = 992 (should be 1024) 
	const int nodes_per_thread = nodes_per_block / 32;	// = 31 (should be 32)
	int thread_array_start_index = nodes_per_thread * thread_id_in_block;
	__shared__ node shared_list[nodes_per_block];
	node* __restrict__ shared_list_thread_pointer = &shared_list[thread_array_start_index];
	int /*open*/   shared_open_it = 0,
		/*closed*/ shared_closed_it = nodes_per_thread - 1;

	const int array_size_diff = (int)(((float)nodes_per_thread / (float)register_list_size) + 0.99);

	//CREATE GLOBAL ARRAY
	int open_list_it = 0, closed_list_it = 0, open_list_size = 2000, closed_list_size = 1400;
	node* __restrict__ open_list = (node*)malloc(2000 * sizeof(node));
	node* __restrict__ closed_list = (node*)malloc(1400 * sizeof(node));

	if (open_list == NULL || closed_list == NULL) {
		printf("Device heap limit to low for lists\n");
		free(open_list);
		free(closed_list);
		return;
	}

	__shared__ bool block_check;
	bool solution_found = false;

	open_list[0] = { start_id, -1, 0, FloatDistance(x, y, destination.x, destination.y) };
	open_list_it = 1;

	if (debug && debug_coord.x == x && debug_coord.y == y) printf("start \n");

	//-----------------------------

	const int max_step_loops = 2800;
	const int max_open_list_size = max_step_loops * 3 + 1;
	const int max_closed_list_size = max_step_loops + 1;
	for (int step_iterator = 0; step_iterator < max_step_loops; ++step_iterator) {
		//~1400 is the nr of iterations it takes for the longest path to be calculated in the complex experiment map
		
		block_check = false;
		if (shared_closed_it - shared_open_it < 6) block_check = true;	//check if 1 or more threads need to move data from shared to global
		if (block_check) {	//transfer shared data to global memory
			if (debug && debug_coord.x == x && debug_coord.y == y) printf("transfering data from shared to global mem\n");

			//GLOBAL READ/WRITE
			memcpy(&open_list[open_list_it], &shared_list_thread_pointer[0], shared_open_it * sizeof(node));
			open_list_it += shared_open_it;
			for (int i = 0; i < nodes_per_thread - 1; ++i) {
				if (i >= (nodes_per_thread - shared_closed_it - 1)) break;
				closed_list[closed_list_it + i] = shared_list_thread_pointer[nodes_per_thread - 1 - i];
			}
			closed_list_it += (nodes_per_thread - shared_closed_it - 1);
			memset(&shared_list_thread_pointer[0], 0, nodes_per_thread * sizeof(node));	//reset shared array
			shared_open_it = 0;
			shared_closed_it = nodes_per_thread - 1;
		}

		//-----------------------------

		//find the next cell to expand
		bool closest_node_in_shared_mem;
		float closest_distance_found = 99999;
		node closest_entry;
		int closest_coord_found = -1;

		//search in shared open list
		for (int slice = 0; slice < array_size_diff; ++slice) {
			for (int i = 0; i < register_list_size; ++i) {
				if ((i + slice * register_list_size) >= shared_open_it) {
					register_list[i].pos = -1;
					register_list[i].backtrack_it = -1;
					register_list[i].steps_from_start = -1;
					register_list[i].est_dist_start_to_dest_via_pos = -1.f;
				}
				else register_list[i] = shared_list_thread_pointer[i + (slice * register_list_size)];
			}

			for (int i = 0; i < register_list_size; ++i) {	//loop over register array
				if ((i + slice * register_list_size) >= shared_open_it) /*break*/ goto search_shared_open_list_end;
				if (register_list[i].pos > 0) {	//if valid node
					if (register_list[i].est_dist_start_to_dest_via_pos </*=*/ closest_distance_found) {	//if closest node
						closest_distance_found = register_list[i].est_dist_start_to_dest_via_pos;
						closest_node_in_shared_mem = true;
						closest_coord_found = i + slice * register_list_size;	//maybe...
						closest_entry = register_list[i];
					}
				}
			}
		}
		search_shared_open_list_end:

		//search in global open list
		int it = 0;
		while (it < open_list_it) {
			//GLOBAL READ/WRITE
			for (int i = 0; i < register_list_size; ++i) {
				if (i >= (open_list_it - it + i)) {
					register_list[i].pos = -2;
					register_list[i].backtrack_it = -2;
					register_list[i].steps_from_start = -2;
					register_list[i].est_dist_start_to_dest_via_pos = -2.f;
				}
				else register_list[i] = open_list[it + i];
			}

			for (int i = 0; i < register_list_size; ++i) {
				if (register_list[i].pos == -258 || register_list[i].pos == -2) /*break*/ goto search_global_open_list_end;
				if (register_list[i].pos > 0 ) {	//if valid node
					if (register_list[i].est_dist_start_to_dest_via_pos </*=*/ closest_distance_found) {	//if closest node
						closest_distance_found = register_list[i].est_dist_start_to_dest_via_pos;
						closest_node_in_shared_mem = false;
						closest_coord_found = it + i;
						closest_entry = register_list[i];
					}
				}
			}
			it += register_list_size;
		}
		search_global_open_list_end:

		//-----------------------------

		if (closest_coord_found == -1) {	//open list is empty and no path to the destination is found, RIP
			((float*)(((char*)device_map.ptr) + y * device_map.pitch))[x] = -3;
			memset(open_list, 0, open_list_it * sizeof(node));
			memset(closed_list, 0, open_list_it * sizeof(node));
			free(open_list);
			free(closed_list);
			return;
		}

		//-----------------------------

		//add the expanded coord to the closed list
		//SHARED READ/WRITE
		shared_list_thread_pointer[shared_closed_it] = closest_entry;
		--shared_closed_it;

		IntPoint2D pos = IDToPos(closest_entry.pos, grid_thread_width);

		if (debug && debug_coord.x == x && debug_coord.y == y) printf("expanded (%d,%d) on step %d \n", pos.x, pos.y, step_iterator);

		//-----------------------------

		if ((pos.x == destination.x) && (pos.y == destination.y)) {	//destination has been found! HYPE
			if (debug && debug_coord.x == x && debug_coord.y == y) printf("destination found! \n");
			solution_found = true;
			break;
		}

		//-----------------------------

		//expand the size of the open and closed list if necessary
		if (step_iterator%30 == 0) {
			block_check = false;
			if ((open_list_size - (open_list_it + shared_open_it)) < 200) block_check = true;
			if (block_check) {	//expand open list
				//GLOBAL READ/WRITE
				node* __restrict__ open_list_new = (node*)malloc(open_list_size * 2 * sizeof(node));
				memcpy(&open_list_new[0], &open_list[0], open_list_size * sizeof(node));
				open_list_size *= 2;
				free(open_list);
				open_list = open_list_new;

				if (debug && debug_coord.x == x && debug_coord.y == y) printf("expanding open list, new size: %d \n", open_list_size);

				if (open_list == NULL) {
					printf("Device heap limit to low for lists (expand)\n");
					memset(open_list, 0, open_list_it * sizeof(node));
					memset(closed_list, 0, open_list_it * sizeof(node));
					free(open_list);
					free(closed_list);
					return;
				}
			}

			block_check = false;
			if ((closed_list_size - (closed_list_it + (nodes_per_thread - 1 - shared_closed_it))) < 200) block_check = true;
			if (block_check) {	//expand closed list
				//GLOBAL READ/WRITE
				node* __restrict__ closed_list_new = (node*)malloc(closed_list_size * 2 * sizeof(node));
				memcpy(&closed_list_new[0], &closed_list[0], closed_list_size * sizeof(node));
				closed_list_size *= 2;
				free(closed_list);
				closed_list = closed_list_new;

				if (debug && debug_coord.x == x && debug_coord.y == y) printf("expanding closed list \n");

				if (closed_list == NULL) {
					printf("Device heap limit to low for lists (expand)\n");
					memset(open_list, 0, open_list_it * sizeof(node));
					memset(closed_list, 0, open_list_it * sizeof(node));
					free(open_list);
					free(closed_list);
					return;
				}
			}
		}

		//-----------------------------

		//add the expanded nodes neighbours to the open list
		IntPoint2D neighbour_coords[4];
		neighbour_coords[0] = { pos.x, pos.y - 1 };	//up
		neighbour_coords[1] = { pos.x + 1, pos.y };	//right
		neighbour_coords[2] = { pos.x, pos.y + 1 };	//down
		neighbour_coords[3] = { pos.x - 1, pos.y };	//left

		integer neighbour_coord_global[4];
		neighbour_coord_global[0] = PosToID({ neighbour_coords[0].x, neighbour_coords[0].y }, grid_thread_width);
		neighbour_coord_global[1] = PosToID({ neighbour_coords[1].x, neighbour_coords[1].y }, grid_thread_width);
		neighbour_coord_global[2] = PosToID({ neighbour_coords[2].x, neighbour_coords[2].y }, grid_thread_width);
		neighbour_coord_global[3] = PosToID({ neighbour_coords[3].x, neighbour_coords[3].y }, grid_thread_width);

		bool neighbour_coord_validity[4];
		neighbour_coord_validity[0] = true;
		neighbour_coord_validity[1] = true;
		neighbour_coord_validity[2] = true;
		neighbour_coord_validity[3] = true;

		if (debug && debug_coord.x == x && debug_coord.y == y) printf("checking neighbours to (%d,%d)<%d>:\n   (%d,%d)<%d>\n   (%d,%d)<%d>\n   (%d,%d)<%d>\n   (%d,%d)<%d> \n",
			pos.x, pos.y, closest_entry.pos, 
			neighbour_coords[0].x, neighbour_coords[0].y, neighbour_coord_global[0], 
			neighbour_coords[1].x, neighbour_coords[1].y, neighbour_coord_global[1],
			neighbour_coords[2].x, neighbour_coords[2].y, neighbour_coord_global[2],
			neighbour_coords[3].x, neighbour_coords[3].y, neighbour_coord_global[3]);

		//-----------------------------

		//use register list for terrain map values
		register_list[0].pos = GetBoolMapValue(dynamic_map, neighbour_coords[0].x, neighbour_coords[0].y);
		register_list[1].pos = GetBoolMapValue(dynamic_map, neighbour_coords[1].x, neighbour_coords[1].y);
		register_list[2].pos = GetBoolMapValue(dynamic_map, neighbour_coords[2].x, neighbour_coords[2].y);
		register_list[3].pos = GetBoolMapValue(dynamic_map, neighbour_coords[3].x, neighbour_coords[3].y);

		//Check the neighbours for invalid positions
		for (int i = 0; i < 4; ++i) {
			if (!(neighbour_coords[i].x <= MAP_X_R) || !(neighbour_coords[i].y <= MAP_Y_R) || !(neighbour_coords[i].x > 0) || !(neighbour_coords[i].y > 0)) {	//coord not in map (FIX UGLINESS!)
				if (debug && debug_coord.x == x && debug_coord.y == y) printf("   neighbour %d failed map bound check \n", i);
				neighbour_coord_validity[i] = false;
			}

			//GLOBAL READ/WRITE
			if (neighbour_coord_validity[i] && !(register_list[i].pos != 0)) {	//coord in terrain
				if (debug && debug_coord.x == x && debug_coord.y == y) printf("   neighbour %d failed terrain check \n", i);
				neighbour_coord_validity[i] = false;
			}
		}

		//-----------------------------

		//Search for id in shared closed_list
		//SHARED READ/WRITE
		for (int slice = 0; slice < array_size_diff; ++slice) {
			for (int i = 0; i < register_list_size; ++i) {
				if ((i + slice * register_list_size) >= (nodes_per_thread - shared_closed_it + 1)) {
					register_list[i].pos = -1;
					register_list[i].backtrack_it = -1;
					register_list[i].steps_from_start = -1;
					register_list[i].est_dist_start_to_dest_via_pos = -1.f;
				}
				else register_list[i] = shared_list_thread_pointer[(shared_closed_it + 1 + i) + (slice * register_list_size)];
			}

			for (int i = 0; i < register_list_size; ++i) {	//loop over register array
				if ((i + register_list_size * slice) + shared_closed_it >= nodes_per_thread - 1) /*break*/ goto shared_closed_list_end;
				if (register_list[i].pos > 0) {	//if valid list node
					for (int j = 0; j < 4; ++j) {	//loop over the 4 neighbours
						if (neighbour_coord_validity[j]) {	//if neighbour is valid	(remove?)
							if (neighbour_coord_global[j] == register_list[i].pos) {	//node already in closed list
								if (debug && debug_coord.x == x && debug_coord.y == y) printf("   neighbour %d failed (shared) closed list check \n", j);
								neighbour_coord_validity[j] = false;
							}
						}
					}
				}
			}
		}
		shared_closed_list_end:

		//Search for id in shared open_list
		//SHARED READ/WRITE
		for (int slice = 0; slice < array_size_diff; ++slice) {
			for (int i = 0; i < register_list_size; ++i) {
				if ((i + slice * register_list_size) >= shared_open_it) {
					register_list[i].pos = -1;
					register_list[i].backtrack_it = -1;
					register_list[i].steps_from_start = -1;
					register_list[i].est_dist_start_to_dest_via_pos = -1.f;
				}
				else register_list[i] = shared_list_thread_pointer[i + (slice * register_list_size)];
			}

			//for (int i = 0; i < (nodes_per_thread - 1); ++i) {	//loop over register array
			for (int i = 0; i < register_list_size; ++i) {	//loop over register array
				if ((i + register_list_size * slice) > (shared_open_it - 1)) /*break*/ goto shared_open_list_end;	//(HINDERS REGISTER ARRAY)
				if (register_list[i].pos > 0) {	//if valid list node
					for (int j = 0; j < 4; ++j) {	//loop over the 4 neighbours
						if (neighbour_coord_validity[j]) {	//if neighbour is valid	(remove?)
							if (neighbour_coord_global[j] == register_list[i].pos) {	//node already in open list
								if (debug && debug_coord.x == x && debug_coord.y == y) printf("   neighbour %d failed (shared) closed list check \n", j);
								neighbour_coord_validity[j] = false;
							}
						}
					}
				}
			}
		}
		shared_open_list_end:

		//Search for id in global closed_list
		it = 0;
		while (it < open_list_it) {
			if ((neighbour_coord_validity[0] + neighbour_coord_validity[1] + neighbour_coord_validity[2] + neighbour_coord_validity[3]) < 1) break;
			//GLOBAL READ/WRITE
			for (int i = 0; i < register_list_size; ++i) {
				if (i >= (closed_list_it - it + i)) {
					register_list[i].pos = -2;
					register_list[i].backtrack_it = -2;
					register_list[i].steps_from_start = -2;
					register_list[i].est_dist_start_to_dest_via_pos = -2.f;
				}
				else register_list[i] = closed_list[it + i];
			}

			for (int i = 0; i < register_list_size; ++i) {	//loop over register array
				if (register_list[i].pos == -258 || register_list[i].pos == -2) /*break*/ goto global_closed_list_end;
				if (register_list[i].pos > 0) {	//if valid list node
					for (int j = 0; j < 4; ++j) {	//loop over the 4 neighbours
						if (neighbour_coord_validity[j]) {	//if neighbour is valid	(remove?)
							if (neighbour_coord_global[j] == register_list[i].pos) {	//node already in closed list
								if (debug && debug_coord.x == x && debug_coord.y == y) printf("   neighbour %d failed (global) closed list check \n", j);
								neighbour_coord_validity[j] = false;
							}
						}
					}
				}
			}
			it += register_list_size;
		}
		global_closed_list_end:

		//Search for id in global open_list
		it = 0;
		while (it < open_list_it) {
			if ((neighbour_coord_validity[0] + neighbour_coord_validity[1] + neighbour_coord_validity[2] + neighbour_coord_validity[3]) < 1) break;
			//GLOBAL READ/WRITE
			for (int i = 0; i < register_list_size; ++i) {
				if (i >= (open_list_it - it + i)) {
					register_list[i].pos = -2;
					register_list[i].backtrack_it = -2;
					register_list[i].steps_from_start = -2;
					register_list[i].est_dist_start_to_dest_via_pos = -2.f;
				}
				else register_list[i] = open_list[it + i];
			}

			for (int i = 0; i < register_list_size; ++i) {	//loop over register array
				if (register_list[i].pos == -258 || register_list[i].pos == -2) /*break*/ goto global_open_list_end;
				if (register_list[i].pos > 0) {	//if valid list node
					for (int j = 0; j < 4; ++j) {	//loop over the 4 neighbours
						if (neighbour_coord_validity[j]) {	//if neighbour is valid	(remove?)
							if (neighbour_coord_global[j] == register_list[i].pos) {	//node already in open list
								if (debug && debug_coord.x == x && debug_coord.y == y) printf("   neighbour %d failed (global) open list check \n", j);
								neighbour_coord_validity[j] = false;
							}
						}
					}
				}
			}
			it += register_list_size;
		}
		global_open_list_end:

		//-----------------------------

		if (debug && debug_coord.x == x && debug_coord.y == y) printf("   nodes to add: ");

		//Add the valid neighbours to the open list
		int new_open_list_entries = 0;
		struct { node node; bool valid = false; } nodes_to_add[4];
		for (int i = 0; i < 4; ++i) {	//loop over the 4 neighbours
			if (neighbour_coord_validity[i]) {	//the neighbour is valid
				node new_list_entry = {
					neighbour_coord_global[i],
					closed_list_it - 1 + (nodes_per_thread - 1 - shared_closed_it),
					closest_entry.steps_from_start + 1,
					closest_entry.steps_from_start + 1 + FloatDistance(neighbour_coords[i].x, neighbour_coords[i].y, destination.x, destination.y)
				};
				nodes_to_add[i].node = new_list_entry;
				nodes_to_add[i].valid = true;
				++new_open_list_entries;

				if (debug && debug_coord.x == x && debug_coord.y == y) printf("(%d, %d, %d, %f) ", new_list_entry.pos, 
					new_list_entry.backtrack_it, new_list_entry.steps_from_start, new_list_entry.est_dist_start_to_dest_via_pos);
			}
		}

		if (debug && debug_coord.x == x && debug_coord.y == y) printf("\n   %d valid neighbours to (%d,%d) \n", new_open_list_entries, pos.x, pos.y);

		//-----------------------------

		//SHARED READ/WRITE
		for (int i = 0; i < 4; ++i) {
			if (nodes_to_add[i].valid) {
				shared_list_thread_pointer[shared_open_it] = nodes_to_add[i].node;
				shared_open_it++;
			}
		}
		
		if (!closest_node_in_shared_mem) {
			//GLOBAL READ/WRITE
			open_list[closest_coord_found].pos = -1;	//mark expanded node as invalid in the open list (global)
		}
		else {
			//SHARED READ/WRITE
			shared_list_thread_pointer[closest_coord_found].pos = -1;	//mark expanded node as invalid in the open list (shared)
		}

		if (debug && debug_coord.x == x && debug_coord.y == y) printf("open_list_it: %d\nclosed_list_it: %d\nshared_open_it: %d\nshared_closed_it: %d\nsize_check_counter: ?\n",
			open_list_it, closed_list_it, shared_open_it, shared_closed_it/*, size_check_counter*/);

	}	//END OF A* LOOP

	__syncthreads();	//unnecessary?

	if (!solution_found) {
		//GLOBAL READ/WRITE
		if(((float*)(((char*)device_map.ptr) + y * device_map.pitch))[x] == 0)	//shit solution, but it works...
			((float*)(((char*)device_map.ptr) + y * device_map.pitch))[x] = -2;
		memset(open_list, 0, open_list_it * sizeof(node));
		memset(closed_list, 0, open_list_it * sizeof(node));
		free(open_list);
		free(closed_list);
		return;
	}

	__syncthreads();

	if (debug && debug_coord.x == x && debug_coord.y == y) printf("transfering data from shared to global mem\n");

	//GLOBAL READ/WRITE
	for (int i = 0; i < nodes_per_thread - 1; ++i) {
		if (i >= (nodes_per_thread - shared_closed_it - 1)) break;
		closed_list[closed_list_it + i] = shared_list_thread_pointer[nodes_per_thread - 1 - i];
	}
	closed_list_it += (nodes_per_thread - shared_closed_it - 1);
	shared_open_it = 0;
	shared_closed_it = nodes_per_thread - 1;

	//--------

	node curr = closed_list[closed_list_it - 1];
	IntPoint2D pos;
	for (int loop_count = 1; loop_count < MAP_SIZE_R + 1; ++loop_count) {
		pos = IDToPos(curr.pos, grid_thread_width);

		if (debug && debug_coord.x == x && debug_coord.y == y) printf("backtrack: printing %d to (%d,%d), node <%d, %d, %d, %f>\n", loop_count, pos.x, pos.y,
			curr.pos, curr.backtrack_it, curr.steps_from_start, curr.est_dist_start_to_dest_via_pos);

		if (curr.steps_from_start < 4) ((float*)(((char*)device_map.ptr) + pos.y * device_map.pitch))[pos.x] = loop_count;

		if (curr.backtrack_it == -1) {
			break;
		}
		curr = closed_list[curr.backtrack_it];
	}

	if (debug && debug_coord.x == x && debug_coord.y == y) printf("backtracking done\n");

	//--------

	memset(open_list, 0, open_list_it * sizeof(node));
	memset(closed_list, 0, open_list_it * sizeof(node));
	free(open_list);
	free(closed_list);
}

__global__ void DeviceAirIMGeneration(IntPoint2D destination, hipPitchedPtr device_map) {
	int block_size = blockDim.x*blockDim.y;
	int grid_size = (gridDim.x * blockDim.x) * (gridDim.y * blockDim.y);

	int id_block = blockIdx.x + blockIdx.y * gridDim.x;
	int original_x = threadIdx.x + blockIdx.x * blockDim.x;
	int original_y = threadIdx.y + blockIdx.y * blockDim.y;
	int original_id = threadIdx.x + id_block * block_size + threadIdx.y * blockDim.x;

	//original
	int start_id = original_id;
	int x = original_x;
	int y = original_y;

	if (x >= MAP_X_R || y >= MAP_Y_R || x < 0 || y < 0) {	//return if start tex is out of bounds 
		return;
	}

	if (destination.x >= MAP_X_R || destination.y >= MAP_Y_R) {	//return if destination is out of bounds
		((float*)(((char*)device_map.ptr) + y * device_map.pitch))[x] = -1;
		if (x == 0 && y == 0) printf("CUDA PRINT: destination out of bounds\n");
		return;
	}

	((float*)(((char*)device_map.ptr) + y * device_map.pitch))[x] = BlockDistance(x, y, destination.x, destination.y) + 1;
}

__global__ void DeviceUpdateDynamicMap(IntPoint2D top_left, IntPoint2D bottom_right, IntPoint2D center, float radius, int new_value, hipPitchedPtr dynamic_map_device_pointer) {
	int local_x = threadIdx.x + blockIdx.x * blockDim.x;
	int local_y = threadIdx.y + blockIdx.y * blockDim.y;

	if (local_x > bottom_right.x || local_y > bottom_right.y) {
		return;
	}

	int global_x = local_x + top_left.x;
	int global_y = local_y + top_left.y;

	FloatPoint2D center_r, corners[4];
	center_r = { ((float)center.x + (0.5 / GRID_DIVISION)) , ((float)center.y + (0.5 / GRID_DIVISION)) };
	corners[0] = { global_x, global_y };
	corners[1] = { global_x + 1, global_y };
	corners[2] = { global_x, global_y + 1 };
	corners[3] = { global_x + 1, global_y + 1 };
	
	float a, b, dist;
	for (int i = 0; i < 4; ++i) {
		a = powf(corners[0].x - center_r.x, 2);
		b = powf(corners[0].y - center_r.y, 2);
		dist = sqrtf(a + b) / GRID_DIVISION;

		if (dist < radius) {
			((bool*)(((char*)dynamic_map_device_pointer.ptr) + global_y * dynamic_map_device_pointer.pitch))[global_x] = new_value;
			return;
		}
	}
}