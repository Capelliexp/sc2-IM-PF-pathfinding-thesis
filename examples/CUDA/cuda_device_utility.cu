#include "hip/hip_runtime.h"
#pragma once

#include "../examples/CUDA/cuda_header.cuh"

//DEVICE SYMBOL VARIABLES (const & global)
__device__ __constant__ UnitInfoDevice* device_unit_lookup;

__device__ float GetFloatMapValue(hipPitchedPtr map, int x, int y) {
	char* ptr = (char*)map.ptr;
	size_t pitch = map.pitch;

	return *((float*)((char*)ptr + y * pitch) + x);
}

//__device__ float GetFloatMapValue(hipPitchedPtr map, int global_id) {
//	int x = (global_id % (MAP_X_R)) + 1;
//	int y = (global_id / (float)(MAP_X_R)) + 1;
//
//	char* ptr = (char*)map.ptr;
//	size_t pitch = map.pitch;
//
//	return *((float*)((char*)ptr + y * pitch) + x);
//}

__device__ bool GetBoolMapValue(hipPitchedPtr map, int x, int y) {
	char* ptr = (char*)map.ptr;
	size_t pitch = map.pitch;

	return *((bool*)((char*)ptr + y * pitch) + x);
}

//__device__ bool GetBoolMapValue(hipPitchedPtr map, int global_id) {
//	int x = (global_id % (MAP_X_R)) + 1;
//	int y = (global_id / (float)(MAP_X_R)) + 1;
//
//	char* ptr = (char*)map.ptr;
//	size_t pitch = map.pitch;
//
//	return *((bool*)((char*)ptr + y * pitch) + x);
//}

/* check if the id is present in the given list. This could possibly be sped up by fetching
many entries at once... */
__device__ int IDInList(int id, node* list, int list_length){
	for (int i = 0; i < list_length; ++i) {
		if (list[i].pos == id) {
			return i;
		}
	}

	return -1;
}

__device__ void SetMapValue(hipPitchedPtr map, int x, int y, float value) {
	char* ptr = (char*)map.ptr;
	size_t pitch = map.pitch;

	float* row = (float*)((char*)ptr + y * pitch);
	row[x] = value;
}


//returnes the distance, not divided for grid sub-division
__device__ float FloatDistance(float posX1, float posY1, float posX2, float posY2) {
	float a = powf(posX2 - posX1, 2);
	float b = powf(posY2 - posY1, 2);
	return sqrtf(a + b)/* / GRID_DIVISION*/;
}

//returnes the distance, not divided for grid sub-division
//__device__ float FloatDistanceFromIDRelative(int ID, IntPoint2D destination) {
//	float a = powf(destination.x - ((ID % MAP_X_R) + 1), 2);
//	float b = powf(destination.y - ((ID / (float)(MAP_X_R)) + 1), 2);
//	return sqrt(a + b);
//}

__device__ int BlockDistance(int posX1, int posY1, int posX2, int posY2) {
	int a = fabsf(posX1 - posX2);
	int b = fabsf(posY1 - posY2);
	return a + b;
}

//__device__ int BlockDistance(int ID, IntPoint2D destination) {
//	int a = fabsf(destination.x - ((ID % MAP_X_R) + 1));
//	int b = fabsf(destination.y - ((ID / (float)(MAP_X_R)) + 1));
//	return a + b;
//}

__device__ int PosToID(IntPoint2D pos) {
	return (pos.x) + ((pos.y) * MAP_X_R);
}

//__device__ int PosToID(IntPoint2D pos) {
//	return (pos.x) + ((pos.y) * (gridDim.x * blockDim.x));
//}

__device__ IntPoint2D IDToPos(int ID) {
	IntPoint2D res;
	res.x = (ID % MAP_X_R);
	res.y = (ID / (float)MAP_X_R);
	return res;
}

//__device__ IntPoint2D IDToPos(int ID) {
//	IntPoint2D res;
//	res.x = (ID % (gridDim.x * blockDim.x));
//	res.y = (ID / (float)(gridDim.x * blockDim.x));
//	return res;
//}