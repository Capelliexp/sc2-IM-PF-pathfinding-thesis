#include "hip/hip_runtime.h"
#pragma once

#include "../examples/CUDA/cuda_header.cuh"

//DEVICE SYMBOL VARIABLES (const & global)
__device__ __constant__ UnitInfoDevice* device_unit_lookup;

__device__ float GetMapValue(hipPitchedPtr map, int x, int y) {
	char* ptr = (char*)map.ptr;
	size_t pitch = map.pitch;

	return *((float*)((char*)ptr + y * pitch) + x);
}

__device__ void SetMapValue(hipPitchedPtr map, int x, int y, float value) {
	char* ptr = (char*)map.ptr;
	size_t pitch = map.pitch;

	float* row = (float*)((char*)ptr + y * pitch);
	row[x] = value;
}

__device__ float FloatDistance(float posX1, float posY1, float posX2, float posY2) {
	float a = powf(posX2 - posX1, 2);
	float b = powf(posY2 - posY1, 2);
	return sqrtf(a + b) / GRID_DIVISION;
}

__device__ int BlockDistance(int posX1, int posY1, int posX2, int posY2) {
	int a = fabsf(posX1 - posX2);
	int b = fabsf(posY1 - posY2);
	return a + b;
}