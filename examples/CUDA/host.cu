#include "CUDA/device.cu"

#include <stdio.h>
#include <string>
#include <iostream>

__host__ bool InitializeCUDA(int* data) {
	int* new_data = Test(data);

	std::cout << data: << std::endl;
	for (int i = 0; i < THREADS_IN_GRID; ++i) {
		std::cout << new_data[i] << ", ";
	}
	std::cout << std::endl;
}

__host__ void Test(int* data) {
	int* new_data = new int[THREADS_IN_GRID];
	int* device_data = 0;

	hipMalloc((void**)&device_data, THREADS_IN_GRID * sizeof(int)));
	hipMemcpy(device_data, data, THREADS_IN_GRID * sizeof(int), hipMemcpyHostToDevice);

	TestDevice<<<BLOCK_AMOUNT, THREADS_PER_BLOCK>>>(device_data);

	hipMemcpy(new_data, device_data, THREADS_IN_GRID * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(device_data);

	return new_data;
}