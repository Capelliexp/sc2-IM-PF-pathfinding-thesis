#pragma once

#include "../examples/CUDA/cuda_header.cuh"
//#include "../examples/CUDA/map_storage.hpp"

#include <stdio.h>
#include <string>
#include <iostream>

__host__ CUDA::CUDA(MapStorage* maps) {
	if (!InitializeCUDA(maps))
		std::cout << "shit b fucked yo" << std::endl;
}

__host__ CUDA::~CUDA() {

}

__host__ void CUDA::Update(clock_t dt_ticks) {
	//float dt = ((float)dt_ticks) / CLOCKS_PER_SEC;	//get dt in seconds

	if (map_storage->update_terrain) {
		TransferDynamicMapToDevice();
		DeleteAllIMs();	//this might be drastic. should search for which require update and delete those...
	}

	FillDeviceUnitArray();
	//run generation of PFs
}

__host__ bool CUDA::InitializeCUDA(MapStorage* maps) {
	std::cout << "Initializing CUDA object" << std::endl;

	map_storage = maps;
	AllocateDeviceMemory();
	TransferStaticMapToDevice();

	return true;
}

__host__ void CUDA::AllocateDeviceMemory(){
	hipMalloc((void**)&static_map_device_pointer, MAP_X * MAP_Y * sizeof(bool));
	hipMalloc((void**)&dynamic_map_device_pointer, MAP_X * MAP_Y * sizeof(bool));
	hipMalloc((void**)&unit_array_device_pointer, 800 * sizeof(UnitStructInDevice));
}

__host__ bool CUDA::FillDeviceUnitArray() {

	return true;
}

__host__ void CUDA::TestRepellingPFGeneration() {
	float* device_map;
	float* new_map = new float[THREADS_IN_GRID];

	hipMalloc((void**)&device_map, THREADS_IN_GRID * sizeof(float));	//allocate space for map on device

	TransferUnitsToDevice();

	TestDevicePFGeneration << <BLOCK_AMOUNT, THREADS_PER_BLOCK >> > (device_map);

	hipMemcpy(new_map, device_map, THREADS_IN_GRID * sizeof(float), hipMemcpyDeviceToHost);	//transfer map to host
	//the memcpy should copy to a host 2D array directly, not like this!

	//hipFree(device_map);	//do not free, space will be used next frame

}

__host__ void CUDA::TestAttractingPFGeneration(float range, bool is_flying, bool can_attack_air, bool can_attack_ground) {
	
}

__host__ void CUDA::TestIMGeneration(sc2::Point2D destination, bool air_route) {
	//TransferDynamicMapToDevice();

	//TestDevice << <BLOCK_AMOUNT, THREADS_PER_BLOCK >> > ();

	//hipMemcpy();
}

__host__ bool CUDA::TransferUnitsToDevice() {
	std::vector<UnitStructInDevice> vec;
	vec.reserve(map_storage->units.size());

	for (auto const& unit : map_storage->units) {

	}

	//transfer to GPU ...

	return true;
}

__host__ bool CUDA::TransferStaticMapToDevice() {

	return true;
}

__host__ bool CUDA::TransferDynamicMapToDevice() {

	return true;
}