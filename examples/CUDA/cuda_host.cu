#pragma once

#include "../examples/CUDA/cuda_header.cuh"
//#include "../examples/CUDA/map_storage.hpp"

#include <stdio.h>
#include <string>
#include <iostream>

__host__ CUDA::CUDA(MapStorage* maps, const sc2::ObservationInterface* observations) :
	map_storage(maps), observation(observations) {

	if (!InitializeCUDA())
		std::cout << "shit b fucked yo" << std::endl;
}

__host__ CUDA::~CUDA() {

}

__host__ void CUDA::PrintGenInfo() {
	SYSTEM_INFO siSysInfo;
	GetSystemInfo(&siSysInfo);
	hipError_t blob;

	int deviceCount = 0, setDevice = 0;
	Check(hipGetDeviceCount(&deviceCount));
	Check(hipSetDevice(setDevice));

	int Rv, Dv;
	hipRuntimeGetVersion(&Rv);
	hipDriverGetVersion(&Dv);

	std::cout <<
		"BLOCK_AMOUNT: " << BLOCK_AMOUNT << std::endl <<
		"THREADS_PER_BLOCK: " << THREADS_PER_BLOCK << std::endl <<
		std::endl << "Available devices: " << deviceCount << std::endl <<
		"Device ID in use: <" << setDevice << ">" << std::endl <<
		"Runtime API version: " << Rv << std::endl <<
		"Driver API version: " << Dv << std::endl;

	std::cout << std::endl;
}

__host__ void CUDA::Update(clock_t dt_ticks) {
	//float dt = ((float)dt_ticks) / CLOCKS_PER_SEC;	//get dt in seconds

	if (map_storage->update_terrain) {
		TransferDynamicMapToDevice();
		//DeleteAllIMs();	//this might be drastic. should search for which require update and delete those...
	}

	FillDeviceUnitArray();
	//run generation of PFs
}

__host__ bool CUDA::InitializeCUDA() {
	std::cout << "Initializing CUDA object" << std::endl;
	
	PrintGenInfo();
	AllocateDeviceMemory();
	TransferStaticMapToDevice();
	CreateDeviceLookup();

	TestLookupTable();

	return true;
}

__host__ void CUDA::AllocateDeviceMemory(){
	//THIS NEEDS TO BE DEFERRED TO AFTER WE KNOW ARRAY SIZES

	hipMalloc((void**)&static_map_device_pointer, MAP_X * MAP_Y * sizeof(bool));
	hipMalloc((void**)&dynamic_map_device_pointer, MAP_X * MAP_Y * sizeof(bool));
	//hipMalloc((void**)&unit_lookup_device_pointer, 156 * sizeof(UnitInfoDevice));
	//hipMalloc((void**)&unit_array_device_pointer, 800 * sizeof(UnitStructInDevice));
}

__host__ void CUDA::CreateDeviceLookup() {

	//host_unit_info[0] = {sc2::UNIT_TYPEID::TERRAN_WIDOWMINEBURROWED, 0, sc2::UnitTypeID::};

	sc2::UnitTypes types = observation->GetUnitTypeData();

	std::cout << "starting unit search" << std::endl;

	int host_iterator = 0;
	for (int i = 1; i < types.size(); ++i) {
		sc2::UnitTypeData data;
		data = types.at(i);
		if (data.unit_type_id == sc2::UNIT_TYPEID::INVALID) continue;

		std::vector<sc2::Weapon> weapons;
		weapons = data.weapons;
		if(weapons.size() > 0 || data.movement_speed > 0){
			//add to "avoid-and-attack" list

			host_unit_info.push_back(UnitInfo());

			sc2::Weapon longest_weapon;
			longest_weapon.range = 0;

			for (auto& const weapon : weapons) {
				if (weapon.range > longest_weapon.range) longest_weapon = weapon;

				if (weapon.type == sc2::Weapon::TargetType::Ground) host_unit_info.at(host_iterator).can_attack_air = false;
				else if (weapon.type == sc2::Weapon::TargetType::Air) host_unit_info.at(host_iterator).can_attack_ground = false;
			}

			host_unit_info.at(host_iterator).range = longest_weapon.range;
			host_unit_info.at(host_iterator).device_id = host_iterator;
			host_unit_info.at(host_iterator).id = data.unit_type_id;
			
			std::vector<sc2::Attribute> att = data.attributes;
			if (std::find(att.begin(), att.end(), sc2::Attribute::Hover) != att.end())
				host_unit_info.at(host_iterator).is_flying = true;

			host_unit_transform.insert({ { data.unit_type_id, host_iterator } });

			++host_iterator;
		}
	}
	std::cout << "Created unit lookup table on host. Nr of elements: " << host_iterator << ". " << std::endl;
	if (host_iterator != 86) 
		std::cout << "This " << (host_iterator < 86 ? "might" : "will") << " end badly..." << std::endl;
	

	for (int i = 0; i < host_iterator; ++i) {
		/*device_unit_lookup_on_host.push_back({ host_unit_info.at(i).range, host_unit_info.at(i).is_flying,
			host_unit_info.at(i).can_attack_air, host_unit_info.at(i).can_attack_ground });*/
		device_unit_lookup_on_host[i] = { host_unit_info.at(i).range, host_unit_info.at(i).is_flying,
			host_unit_info.at(i).can_attack_air, host_unit_info.at(i).can_attack_ground };
	}

	std::cout << "device_unit_lookup array filled on host" << std::endl;
	TransferSymbolsToDevice();
	std::cout << "device_unit_lookup array copied to device (i think)" << std::endl;
}

__host__ bool CUDA::FillDeviceUnitArray() {

	return true;
}

__host__ void CUDA::TestLookupTable(){

	float* device_write_data;
	hipMalloc((void**)&device_write_data, 156 * sizeof(float));

	TestDeviceLookupUsage<<<1, 156>>>(/*unit_lookup_device_pointer, */device_write_data);

	float* device_return_data = new float[156];
	hipMemcpy(device_return_data, device_write_data, 156 * sizeof(float), hipMemcpyDeviceToHost);

	std::cout << "TestLookupTable() device return data:" << std::endl;
	for (int i = 0; i < 156; ++i) {
		std::cout << device_return_data[i] << ", ";
	}
}

__host__ void CUDA::TestRepellingPFGeneration() {
	float* device_map;
	float* new_map = new float[THREADS_IN_GRID];

	hipMalloc((void**)&device_map, THREADS_IN_GRID * sizeof(float));	//allocate space for map on device

	TransferUnitsToDevice();

	TestDevicePFGeneration << <BLOCK_AMOUNT, THREADS_PER_BLOCK >> > (device_map);

	hipMemcpy(new_map, device_map, THREADS_IN_GRID * sizeof(float), hipMemcpyDeviceToHost);	//transfer map to host
	//the memcpy should copy to a host 2D array directly, not like this!

	//hipFree(device_map);	//do not free, space will be used next frame

}

__host__ void CUDA::TestAttractingPFGeneration(float range, bool is_flying, bool can_attack_air, bool can_attack_ground) {
	
}

__host__ void CUDA::TestIMGeneration(sc2::Point2D destination, bool air_route) {
	//TransferDynamicMapToDevice();

	//TestDevice << <BLOCK_AMOUNT, THREADS_PER_BLOCK >> > ();

	//hipMemcpy();
}

__host__ bool CUDA::TransferUnitsToDevice() {
	//std::vector<UnitStructInDevice> vec;
	//vec.reserve(map_storage->units.size());


	for (auto const& unit : map_storage->units) {

	}

	//transfer to GPU ...

	return true;
}

__host__ bool CUDA::TransferStaticMapToDevice() {

	return true;
}

__host__ bool CUDA::TransferDynamicMapToDevice() {

	return true;
}

__host__ void CUDA::Check(hipError_t blob, std::string location, bool print_res){
	if (blob != hipSuccess) {
		std::cout << "CUDA ERROR: (" << location << ") " << hipGetErrorString(blob) << std::endl;
		blob = hipDeviceReset(); //might be drastic...
	}
	else if (print_res) {
		std::cout << "CUDA STATUS (" << location << ") SUCESS: " << hipGetErrorString(blob) << std::endl;
	}
}