#include "hip/hip_runtime.h"
#pragma once

#include "../examples/CUDA/cuda_header.cuh"

#include <stdio.h>
#include <string>
#include <iostream>
#include <sstream>
#include <fstream>
#include <cmath>
#include <windows.h>

#include "../examples/CUDA/cuda_device_functionality.cu"
#include "../examples/CUDA/cuda_device_tests.cu"

__host__ CUDA::CUDA(){	
}

__host__ CUDA::~CUDA() {

}

__host__ void CUDA::PrintGenInfo() {
	SYSTEM_INFO siSysInfo;
	GetSystemInfo(&siSysInfo);

	int deviceCount = 0, setDevice = 0;
	Check(hipGetDeviceCount(&deviceCount));
	Check(hipSetDevice(setDevice));

	int Rv, Dv;
	hipRuntimeGetVersion(&Rv);
	hipDriverGetVersion(&Dv);

	std::cout <<
		"MAP_DIM: <" << MAP_X << ", " << MAP_Y << "> " << std::endl <<
		"GRID_DIVISION: " << GRID_DIVISION << std::endl <<
		std::endl << "Available devices: " << deviceCount << std::endl <<
		"Device ID in use: <" << setDevice << ">" << std::endl <<
		"Runtime API version: " << Rv << std::endl <<
		"Driver API version: " << Dv << std::endl;

	std::cout << std::endl;
}

__host__ void CUDA::Update(clock_t dt_ticks, sc2::Units units, float ground_avoidance_PF[][MAP_Y_R][1], float air_avoidance_PF[][MAP_Y_R][1]) {
	//float dt = ((float)dt_ticks) / CLOCKS_PER_SEC;	//get dt in seconds
	
	PopErrorsCheck("CUDA Update pre");	//run first

	//if (map_storage->update_terrain) {
	//	TransferDynamicMapToDevice();
	//	//DeleteAllIMs();	//this might be drastic. should search for which require update and delete those...
	//}


	FillDeviceUnitArray(units);
	TransferUnitsToDevice();
	RepellingPFGeneration(ground_avoidance_PF, air_avoidance_PF);

	PopErrorsCheck("CUDA Update post");	//run last

	//run generation of PFs
}

__host__ void CUDA::InitializeCUDA(const sc2::ObservationInterface* observations, sc2::DebugInterface* debug, sc2::ActionInterface* actions){
	std::cout << "Initializing CUDA object" << std::endl;
	
	size_t size;
	hipDeviceGetLimit(&size, hipLimitMallocHeapSize);
	std::cout << "CUDA base heap size: " << size << std::endl;

	hipDeviceSetLimit(hipLimitMallocHeapSize, size*128);
	hipDeviceGetLimit(&size, hipLimitMallocHeapSize);
	std::cout << "CUDA new heap size: " << size << std::endl;

	this->observation = observations;
	this->debug = debug;
	this->actions = actions;
	this->actions_feature_layer = actions_feature_layer;

	dim_block_high = { 32, 32, 1 };
	unsigned int x1 = (unsigned int)(ceil((MAP_X_R - 1) / (float)dim_block_high.x) + 0.5);
	unsigned int y1 = (unsigned int)(ceil((MAP_Y_R - 1) / (float)dim_block_high.y) + 0.5);
	dim_grid_high = { x1, y1, 1 };
	threads_in_grid_high = (dim_block_high.x * dim_block_high.y) * (dim_grid_high.x * dim_grid_high.y);

	dim_block_low = { 8, 4, 1 };
	unsigned int x2 = (unsigned int)(ceil((MAP_X_R - 1) / (float)dim_block_low.x) + 0.5);
	unsigned int y2 = (unsigned int)(ceil((MAP_Y_R - 1) / (float)dim_block_low.y) + 0.5);
	dim_grid_low = { x2, y2, 1 };
	threads_in_grid_low = (dim_block_low.x * dim_block_low.y) * (dim_grid_low.x * dim_grid_low.y);

	unit_list_max_length = 800;
	unit_type_attracting_pf_pointers.reserve(100);
	im_pointers.reserve(100);

	PopErrorsCheck("CUDA Initialization base");

	//analysis
	PrintGenInfo();

	//device_malloc
	AllocateDeviceMemory();

	PopErrorsCheck("CUDA Initialization malloc");

	//IMGeneration(IntPoint2D{ 18, 29 }, false);

	//Check(hipPeekAtLastError(), "init check 7", true);
}

__host__ void CUDA::HostTransfer(sc2::Units units) {
	//host_transfer
	TransferStaticMapToHost();
	FillDeviceUnitArray(units);

	Check(hipPeekAtLastError(), "init check 4", true);
}

__host__ void CUDA::DeviceTransfer(bool dynamic_terrain[][MAP_Y_R][1]) {
	//device_transfer
	TransferDynamicMapToDevice(dynamic_terrain);
	TransferUnitLookupToDevice();
	TransferUnitsToDevice();

	Check(hipPeekAtLastError(), "init check 5", true);
}

__host__ void CUDA::Tests(float ground_avoidance_PF[][MAP_Y_R][1], float air_avoidance_PF[][MAP_Y_R][1]) {
	//tests
	//TestLookupTable();
	Check(hipPeekAtLastError(), "init check 6a", true);

	//Test3DArrayUsage();
	Check(hipPeekAtLastError(), "init check 6b", true);

	RepellingPFGeneration(ground_avoidance_PF, air_avoidance_PF);
	Check(hipPeekAtLastError(), "init check 6c", true);
}

__host__ const sc2::ObservationInterface* CUDA::GetObservation(){
	return observation;
}

__host__ sc2::DebugInterface* CUDA::GetDebug(){
	return debug;
}

__host__ sc2::ActionInterface* CUDA::GetAction(){
	return actions;
}

__host__ sc2::ActionFeatureLayerInterface* CUDA::GetActionFeature(){
	return actions_feature_layer;
}

__host__ void CUDA::CreateUnitLookupOnHost(std::string file){
	sc2::UnitTypes types = observation->GetUnitTypeData();

	int host_iterator = 0;
	for (int i = 1; i < types.size(); ++i) {
		sc2::UnitTypeData data = types.at(i);
		//Check for units that are not considered valid.
		std::string str = sc2::UnitTypeToName(data.unit_type_id.ToType());
		if (str.find("PROTOSS") != std::string::npos || str.find("TERRAN") != std::string::npos || str.find("ZERG") != std::string::npos) {
			std::vector<sc2::Attribute> att = data.attributes;
			if (data.weapons.size() == 0 && std::find(att.begin(), att.end(), sc2::Attribute::Structure) != att.end()) continue;
			host_unit_info.push_back(UnitInfo());

			std::vector<sc2::Weapon> weapons = data.weapons;
			int longest_weapon_range;
			longest_weapon_range = 0;
			for (auto& const weapon : weapons) {
				if (weapon.range > longest_weapon_range)
					longest_weapon_range = weapon.range;

				if (weapon.type == sc2::Weapon::TargetType::Ground)
					host_unit_info.at(host_iterator).can_attack_air = false;
				else if (weapon.type == sc2::Weapon::TargetType::Air)
					host_unit_info.at(host_iterator).can_attack_ground = false;
			}
			host_unit_info.at(host_iterator).range = longest_weapon_range;
			host_unit_info.at(host_iterator).device_id = host_iterator;
			host_unit_info.at(host_iterator).id = data.unit_type_id;


			if (std::find(att.begin(), att.end(), sc2::Attribute::Hover) != att.end())
				host_unit_info.at(host_iterator).is_flying = true;

			host_unit_transform.insert({ { data.unit_type_id, host_iterator } });

			/*
			//failed attempt att doing it the easy way... :(

			debug->DebugCreateUnit(data.unit_type_id, sc2::Point2D(0,0));
			debug->SendDebug();
			actions->SendActions();
			actions_feature_layer->SendActions();
			sc2::Units u = observation->GetUnits();
			for (int i = 0; i < u.size() + 1; ++i) {
				if (i == u.size()) {
					std::cout << "FAILED to get radius of unit: ''" << data.unit_type_id << "''" << std::endl;
					for (int j = 0; j < u.size(); ++j) debug->DebugKillUnit(u.at(j));
					break;
				}
				if (u.at(i)->unit_type == data.unit_type_id) {
					host_unit_info.at(host_iterator).radius = u.at(i)->radius;
					debug->DebugKillUnit(u.at(i));
					break;
				}
			}
			*/

			++host_iterator;
		}
	}
	PrintUnitInfoToFile(file);
	std::cout << "Created unit data table on host. Nr of elements: " << host_iterator << ". " << std::endl;

	for (int i = 0; i < host_unit_info.size(); ++i) {
		device_unit_lookup_on_host.push_back({ host_unit_info.at(i).range, host_unit_info.at(i).radius,
			host_unit_info.at(i).is_flying, host_unit_info.at(i).can_attack_air,
			host_unit_info.at(i).can_attack_ground });
	}
	std::cout << std::endl;
	std::cout << "device_unit_lookup array filled on host" << std::endl;
}

__host__ void CUDA::TransferStaticMapToHost(){}

__host__ void CUDA::TransferUnitLookupToDevice(){
	Check(hipMemcpy(unit_lookup_device_pointer, device_unit_lookup_on_host.data(), device_unit_lookup_on_host.size() * sizeof(UnitInfoDevice), hipMemcpyHostToDevice), "lookup_memcpy");
	Check(hipMemcpyToSymbol(HIP_SYMBOL(device_unit_lookup), &unit_lookup_device_pointer, sizeof(UnitInfoDevice*)), "lookup_symbol_memcpy");
	std::cout << "device_unit_lookup array transfered to device" << std::endl;
}

__host__ void CUDA::AllocateDeviceMemory(){
	//hipMalloc3D(&static_map_device_pointer, hipExtent{ MAP_X_R * sizeof(bool), MAP_Y_R, 1 });	//static map
	hipMalloc3D(&dynamic_map_device_pointer, hipExtent{ MAP_X_R * sizeof(bool), MAP_Y_R, 1 });	//dynamic map
	hipMalloc(&unit_lookup_device_pointer, device_unit_lookup_on_host.size() * sizeof(UnitInfoDevice));	//lookup table (global on device)
	hipMalloc((void**)&device_unit_list_pointer, unit_list_max_length * sizeof(Entity));	//unit list (might extend size during runtime)
	hipMalloc3D(&repelling_pf_ground_map_pointer, hipExtent{ MAP_X_R * sizeof(float), MAP_Y_R, 1 });	//repelling on ground
	hipMalloc3D(&repelling_pf_air_map_pointer, hipExtent{ MAP_X_R * sizeof(float), MAP_Y_R, 1 });	//repelling in air
	Check(hipMalloc((void**)&global_memory_im_list_storage, 256000000 * sizeof(list_double_entry)), "big AF allocation", true);	//big AF list for A* open/closed list

	Check(hipPeekAtLastError(), "cuda allocation peek", true);
}

__host__ void CUDA::FillDeviceUnitArray(sc2::Units units) {
	host_unit_list.clear();
	host_unit_list.resize(units.size());

	//int device_list_length = map_storage->units.size();
	int device_list_length = 0;
	for (const sc2::Unit* unit : units) {
		std::unordered_map<sc2::UNIT_TYPEID, unsigned int>::const_iterator it = host_unit_transform.find(unit->unit_type);
		if (it == host_unit_transform.end()) {
			host_unit_list.resize(host_unit_list.size() - 1);
			std::cout << "WARNING: invalid entity in map_storage unit vector" << std::endl;
			continue;
		}

		host_unit_list.at(device_list_length).id = it->second;
		host_unit_list.at(device_list_length).pos = { unit->pos.x * GRID_DIVISION, unit->pos.y * GRID_DIVISION };
		switch (unit->alliance)
		{
		case sc2::Unit::Alliance::Self:
			host_unit_list.at(device_list_length).enemy = false;
			break;
		case sc2::Unit::Alliance::Ally:
			host_unit_list.at(device_list_length).enemy = false;
			break;
		case sc2::Unit::Alliance::Neutral:
			host_unit_list.at(device_list_length).enemy = false;
				break;
		case sc2::Unit::Alliance::Enemy:
			host_unit_list.at(device_list_length).enemy = true;
			break;
		default:
			host_unit_list.at(device_list_length).enemy = false;
			break;
		}

		device_list_length++;
	}	
}

__host__ void CUDA::TransferUnitsToDevice() {

	if (host_unit_list.size() > unit_list_max_length) {
		std::cout << "WARNING: too many units! Increase allocation size, overflow discarded" << std::endl;
	}

	Check(hipMemcpy(device_unit_list_pointer, host_unit_list.data(), 
		min(host_unit_list.size(), unit_list_max_length) * sizeof(Entity),
		hipMemcpyHostToDevice),
		"TransferUnitsToDevice");
}

__host__ void CUDA::TransferDynamicMapToDevice(bool dynamic_terrain[][MAP_Y_R][1]) {
	hipMemcpy3DParms par = { 0 };
	par.srcPtr = make_hipPitchedPtr((void*)dynamic_terrain, MAP_X_R * sizeof(bool), MAP_X_R, MAP_Y_R);
	par.dstPtr.ptr = dynamic_map_device_pointer.ptr;
	par.dstPtr.pitch = dynamic_map_device_pointer.pitch;
	par.dstPtr.xsize = MAP_X_R;
	par.dstPtr.ysize = MAP_Y_R;
	par.extent.width = MAP_X_R * sizeof(bool);
	par.extent.height = MAP_Y_R;
	par.extent.depth = 1;
	par.kind = hipMemcpyHostToDevice;

	Check(hipMemcpy3D(&par), "Dynamic map transfer");
}

/*KERNAL LAUNCHES START*/

__host__ void CUDA::RepellingPFGeneration(float ground_avoidance_PF[][MAP_Y_R][1], float air_avoidance_PF[][MAP_Y_R][1]) {
	DeviceRepellingPFGeneration<<<dim_grid_high, dim_block_high, (host_unit_list.size() * sizeof(Entity))>>>
		(device_unit_list_pointer, host_unit_list.size(), repelling_pf_ground_map_pointer, repelling_pf_air_map_pointer);

	hipMemcpy3DParms par = { 0 };
	par.srcPtr.ptr = repelling_pf_ground_map_pointer.ptr;
	par.srcPtr.pitch = repelling_pf_ground_map_pointer.pitch;
	par.srcPtr.xsize = MAP_X_R;
	par.srcPtr.ysize = MAP_Y_R;
	par.dstPtr.ptr = ground_avoidance_PF;
	par.dstPtr.pitch = MAP_X_R * sizeof(float);
	par.dstPtr.xsize = MAP_X_R;
	par.dstPtr.ysize = MAP_Y_R;
	par.extent.width = MAP_X_R * sizeof(float);
	par.extent.height = MAP_Y_R;
	par.extent.depth = 1;
	par.kind = hipMemcpyDeviceToHost;

	Check(hipPeekAtLastError(), "PF generation peek 1", true);
	Check(hipDeviceSynchronize(), "PF generation sync", true);
	Check(hipPeekAtLastError(), "PF generation peek 2", true);

	Check(hipMemcpy3D(&par), "ground PF memcpy3D");

	par.srcPtr.ptr = repelling_pf_air_map_pointer.ptr;
	par.srcPtr.pitch = repelling_pf_air_map_pointer.pitch;
	par.dstPtr.ptr = air_avoidance_PF;

	Check(hipMemcpy3D(&par), "air PF memcpy3D");

	//Check(hipDeviceSynchronize());
}

__host__ void CUDA::AttractingPFGeneration(int owner_type_id, float map[][MAP_Y_R][1]){

	hipPitchedPtr device_map;
	hipMalloc3D(&device_map, hipExtent{ MAP_X_R * sizeof(float), MAP_Y_R, 1 });

	DeviceAttractingPFGeneration << <dim_grid_high, dim_block_high, (host_unit_list.size() * sizeof(Entity)) >> >
		(device_unit_list_pointer, host_unit_list.size(), owner_type_id, device_map);

	hipMemcpy3DParms par = { 0 };
	par.srcPtr.ptr = device_map.ptr;
	par.srcPtr.pitch = device_map.pitch;
	par.srcPtr.xsize = MAP_X_R;
	par.srcPtr.ysize = MAP_Y_R;
	par.dstPtr.ptr = map;
	par.dstPtr.pitch = MAP_X_R * sizeof(float);
	par.dstPtr.xsize = MAP_X_R;
	par.dstPtr.ysize = MAP_Y_R;
	par.extent.width = MAP_X_R * sizeof(float);
	par.extent.height = MAP_Y_R;
	par.extent.depth = 1;
	par.kind = hipMemcpyDeviceToHost;

	Check(hipPeekAtLastError(), "PF generation peek 1", true);
	Check(hipDeviceSynchronize(), "PF generation sync", true);
	Check(hipPeekAtLastError(), "PF generation peek 2", true);

	Check(hipMemcpy3D(&par), "ground PF memcpy3D");

	//Check(hipDeviceSynchronize());
}

__host__ void CUDA::IMGeneration(IntPoint2D destination, float map[][MAP_Y_R][1], bool air_path) {

	hipPitchedPtr device_map;
	hipMalloc3D(&device_map, hipExtent{ MAP_X_R * sizeof(float), MAP_Y_R, 1 });

	IntPoint2D destination_R = {destination.x * GRID_DIVISION, destination.y * GRID_DIVISION};

	if (!air_path) {
		DeviceGroundIMGeneration <<<dim_grid_low, dim_block_low>>>
			(destination_R, device_map, dynamic_map_device_pointer, global_memory_im_list_storage);
	}
	else {
		DeviceAirIMGeneration <<<dim_grid_low, dim_block_high>>> (destination_R, device_map);
	}

	Check(hipPeekAtLastError(), "IM generation peek 1", true);
	Check(hipDeviceSynchronize(), "IM generation sync", true);

	PopErrorsCheck("IMGen");

	hipMemcpy3DParms par = { 0 };
	par.srcPtr.ptr = device_map.ptr;
	par.srcPtr.pitch = device_map.pitch;
	par.srcPtr.xsize = MAP_X_R;
	par.srcPtr.ysize = MAP_Y_R;
	par.dstPtr.ptr = map;
	par.dstPtr.pitch = MAP_X_R * sizeof(float);
	par.dstPtr.xsize = MAP_X_R;
	par.dstPtr.ysize = MAP_Y_R;
	par.extent.width = MAP_X_R * sizeof(float);
	par.extent.height = MAP_Y_R;
	par.extent.depth = 1;
	par.kind = hipMemcpyDeviceToHost;

	Check(hipMemcpy3D(&par), "IM memcpy3D", true);

	Check(hipDeviceSynchronize(), "IM print sync", true);
}

__host__ void CUDA::UpdateDynamicMap(IntPoint2D center, float radius, int value) {
	IntPoint2D top_left = { center.x - radius - 1, center.y - radius - 1};
	IntPoint2D bottom_right = { center.x + radius + 1, center.y + radius + 1};

	DeviceUpdateDynamicMap <<< {((bottom_right.x - top_left.x) / dim_block_high.x) + 1, ((bottom_right.y - top_left.y) / dim_block_high.y) + 1, 1},
		dim_block_high >> > (top_left, bottom_right, center, radius, value, dynamic_map_device_pointer);
}

__host__ void CUDA::TestLookupTable() {
	int table_length = device_unit_lookup_on_host.size();

	float* write_data_d;
	hipMalloc((void**)&write_data_d, table_length * sizeof(float));

	TestDeviceLookupUsage << <1, table_length >> > (write_data_d);

	float* return_data = new float[table_length];
	hipMemcpy(return_data, write_data_d, table_length * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < table_length; ++i) {
		if (std::abs(return_data[i] - device_unit_lookup_on_host[i].range) > 0.01) {
			std::cout << "lookup table test FAILED" << std::endl;
			delete return_data;
			hipFree(write_data_d);
			return;
		}
	}
	std::cout << "lookup table test SUCCESS" << std::endl;

	delete return_data;
	hipFree(write_data_d);
}

__host__ void CUDA::Test3DArrayUsage() {
	hipPitchedPtr device_map;
	Check(hipMalloc3D(&device_map, hipExtent{ MAP_X_R * sizeof(float), MAP_Y_R, 1 }), "PFGeneration malloc3D");

	TransferUnitsToDevice();	//unnecessary for the test

	TestDevice3DArrayUsage << <1, MAP_SIZE_R, (host_unit_list.size() * sizeof(Entity)) >> >
		(device_unit_list_pointer, host_unit_list.size(), device_map);

	float *return_data = new float(MAP_X_R * MAP_Y_R);
	hipMemcpy3DParms par = { 0 };
	par.srcPtr.ptr = device_map.ptr;
	par.srcPtr.pitch = device_map.pitch;
	par.srcPtr.xsize = MAP_X_R;
	par.srcPtr.ysize = MAP_Y_R;
	par.dstPtr.ptr = return_data;
	par.dstPtr.pitch = MAP_X_R * sizeof(float);
	par.dstPtr.xsize = MAP_X_R;
	par.dstPtr.ysize = MAP_Y_R;
	par.extent.width = MAP_X_R * sizeof(float);
	par.extent.height = MAP_Y_R;
	par.extent.depth = 1;
	par.kind = hipMemcpyDeviceToHost;

	Check(hipMemcpy3D(&par), "memcpy3D");

	Check(hipDeviceSynchronize());

	//check
	int it = 0;
	for (int i = 0; i < MAP_X_R; ++i) {
		for (int j = 0; j < MAP_Y_R; ++j) {
			if (return_data[i + j * MAP_X_R] != i * MAP_X_R + j) {
				std::cout << "3D Array Usage test FAILED" << std::endl;
				return;
			}
		}
	}
	std::cout << "3D Array Usage test SUCCESS" << std::endl;

	//hipFree(device_map);	//do not free, space will be used next frame
}

__host__ void CUDA::TestAttractingPFGeneration() {

}

__host__ void CUDA::TestIMGeneration(sc2::Point2D destination, bool air_route) {
	//TransferDynamicMapToDevice();

	//TestDevice << <BLOCK_AMOUNT, THREADS_PER_BLOCK >> > ();

	//hipMemcpy();
}

/*KERNAL LAUNCHES END*/

__host__ void CUDA::Check(hipError_t blob, std::string location, bool print_res){
	if (blob != hipSuccess) {
		std::cout << "CUDA ERROR: (" << location << ") " << hipGetErrorString(blob) << std::endl;
		blob = hipDeviceReset(); //might be drastic...
	}
	else if (print_res) {
		std::cout << "CUDA STATUS (" << location << ") SUCESS: " << hipGetErrorString(blob) << std::endl;
	}
}

__host__ void CUDA::PopErrorsCheck(std::string location) {
	int it = 0;
	while (hipPeekAtLastError() != hipSuccess) {
		Check(hipGetLastError(), ("error pop repeat <" + location + "> " + std::to_string(it)));
		++it;
	}
}

__host__ void CUDA::PrintUnitInfoToFile(std::string filename) {
	std::stringstream str(std::stringstream::out);
	str << "UnitID, DeviceID, Radius, WeaponRange, CanAttackGround, CanAttackAir, IsFlying" << std::endl;

	for (UnitInfo unit : this->host_unit_info) {
		str << unit.id << "," << unit.device_id << ","
			<< unit.radius << "," << unit.range << ","
			<< unit.can_attack_ground << "," << unit.can_attack_air << ","
			<< unit.is_flying << std::endl;
	}

	std::ofstream file;
	file.open(filename);
	file.write(str.str().c_str(), str.str().length());
	file.close();
}

__host__ void CUDA::ReadUnitInfoFromFile(std::string filename) {
	this->host_unit_info.clear();
	std::ifstream inFile(filename);
	std::string line;
	std::getline(inFile, line);	//Remove the first line
	int host_iterator = 0;
	while (std::getline(inFile, line)) {
		UnitInfo unit;
		int pos = line.find(",");
		unit.id = std::stoi(line.substr(0, pos));
		line.erase(0, pos + 1);
		
		pos = line.find(",");
		unit.device_id = std::stoi(line.substr(0, pos));
		line.erase(0, pos + 1);

		pos = line.find(",");
		unit.radius = std::stof(line.substr(0, pos));
		line.erase(0, pos + 1);

		pos = line.find(",");
		unit.range = std::stof(line.substr(0, pos));
		line.erase(0, pos + 1);

		pos = line.find(",");
		unit.can_attack_ground = std::stoi(line.substr(0, pos));
		line.erase(0, pos + 1);

		pos = line.find(",");
		unit.can_attack_air = std::stoi(line.substr(0, pos));
		line.erase(0, pos + 1);

		pos = line.find(",");
		unit.is_flying = std::stoi(line.substr(0, pos));
		line.erase(0, pos + 1);

		this->host_unit_info.push_back(unit);
		this->host_unit_transform.insert({ { sc2::UNIT_TYPEID(unit.id), host_iterator } });
		++host_iterator;
	}
}

__host__ std::vector<int> CUDA::GetUnitsID() {
	std::vector<int> unit_IDs;
	for (UnitInfo unit : host_unit_info) {
		unit_IDs.push_back(unit.id);
	}
	return unit_IDs;
}

__host__ void CUDA::SetRadiusForUnits(std::vector<float> radius) {
	for (int i = 0; i < radius.size(); ++i) {
		host_unit_info[i].radius = radius[i];
	}
	PrintUnitInfoToFile("unitInfo.txt");
}

__host__ void CUDA::SetIsFlyingForUnits(std::vector<bool> is_flying) {
	for (int i = 0; i < is_flying.size(); ++i) {
		host_unit_info[i].is_flying = is_flying[i];
	}
	PrintUnitInfoToFile("unitInfo.txt");
}

__host__ int CUDA::GetPosOFUnitInHostUnitVec(sc2::UNIT_TYPEID typeID) {
	return host_unit_transform.at(typeID);
}

__host__ int CUDA::GetSizeOfUnitInfoList() {
	return host_unit_info.size();
}
